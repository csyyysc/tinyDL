#include <CudaHelper.cuh>
#include <iostream>

void checkCudaError(hipError_t err, const std::string &msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCudaMallocError(const std::string &entity, hipError_t err, int device_id) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl;
        std::cerr << "CUDA error: Failed to allocate memory for " << entity << " on device: " << device_id << " - "
                  << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA malloc failed");
    }
}