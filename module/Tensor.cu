#include "CudaHelper.cuh"
#include "Tensor.cuh"
#include <iostream>
#include <stdexcept>

Tensor::Tensor(const std::vector<int> &shape_vec, bool requires_grad, int dev_id)
    : shape_vec(shape_vec), requires_grad(requires_grad), device_id(dev_id), is_device(false) {
    strides = calculate_strides(shape_vec);
    num_of_elements = calculate_size(shape_vec);

    hipError_t cuda_err;
    hipSetDevice(device_id);
    cuda_err = hipMalloc((void **)&data, num_of_elements * sizeof(float));
    checkCudaMallocError("data", cuda_err, device_id);

    if (requires_grad) {
        cuda_err = hipMalloc((void **)&grad, num_of_elements * sizeof(float));
        checkCudaMallocError("grad", cuda_err, device_id);
    }
}

// @todo: Tensor constructor to create a Tensor from existing host data (copies data to device)

// @todo: Tensor constructor to create a Tensor from existing device data

void Tensor::zero_grad(hipStream_t stream) {
    if (requires_grad && grad) {
        hipError_t cuda_err;
        hipSetDevice(device_id);
        cuda_err = hipMemsetAsync(grad.get(), 0, num_of_elements * sizeof(float), stream);
        checkCudaMallocError("grad", cuda_err, device_id);
    }
}

void Tensor::backward(hipStream_t stream) {
    if (!requires_grad)
        return;

    hipError_t cuda_err;
    hipSetDevice(device_id);
    cuda_err = hipMemsetAsync(grad.get(), 1, num_of_elements * sizeof(float), stream);
    checkCudaMallocError("grad", cuda_err, device_id);

    for (const auto &dep : dependencies) {
        dep.backward_fn(grad.get(), stream);
    }

    // void backward() {
    //     if (!requires_grad)
    //         return;
    //     std::fill(grad, grad + size(), 1.0f);
    //     for (auto &dep : dependencies) {
    //         dep.backward_fn(grad);
}
